#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <sstream>

__device__
double nextLGP_m_eq0(int l, double x, double p_0, double p_1) {
// x = cos(theta)
  return __dadd_rd(__dmul_rd(__dmul_rd(p_1, __ddiv_rd(2*l-1, l)),x), __dmul_rd(p_0, __ddiv_rd(l-1, l))*-1); 
}

__device__
double nextDp_m_eq_0(int l, double lgp_mp) {
   return __dmul_rd(-1*lgp_mp, __dsqrt_rd(l*(l+1)/2)); 
}

__device__
double nextDp_m_eq_1(int l, double lgp_mn, double lgp_mp) {
  return __dmul_rd(0.5, __dadd_rd(__dmul_rd(__dsqrt_rd(2*l*(l+1)), lgp_mn),-1 * __dmul_rd(__dsqrt_rd((l-1)*(l+2)), lgp_mp) ));
}

__device__
double nextDp_m_l(int m, int l, double lgp_mn, double lgp_mp) {
  return __dmul_rd(0.5, __dadd_rd(__dmul_rd(__dsqrt_rd((l+m)*(l-m+1)), lgp_mn), -1*__dmul_rd(__dsqrt_rd((l-m)*(l+m+1)), lgp_mp)));
}

__device__
double calculateLGP_m_eq_l(int mode) {
  double lgp=1;
  for(int k=1; k<=abs(mode); k++) {
    lgp *= __ddiv_ru((double)2*k-1, (double)2*k);
  }
  
  return __dsqrt_rd(__dmul_rd(2, lgp));
}

__device__ 
double calculateLGP_mp1_eq_l(int m, double x, double lgp_m_eq_l) {
  int mode = abs(m);
  return __dmul_rd(__dmul_rd(lgp_m_eq_l, __dsqrt_rd(2*mode+1)),x); 
}

__device__
double calculateLGP_m_l(int mode, int degree, double theta, double lgp_0, double lgp_1) {
  int m = abs(mode);
  return  __ddiv_rd(__dadd_rd(__dmul_rd(2*degree-1, __dmul_rd(cos(theta), lgp_1)), __dmul_rd(-1 * lgp_0, __dsqrt_rd((degree-1)*(degree-1) - m*m))), __dsqrt_rd((degree*degree) - (m*m)));
}

__device__
double scaleBySine(int m, double lgp, double theta) {
  double reg1 = sin(theta);
  for(int r=0; r<abs(m); r++)
    lgp = __dmul_rd(lgp,reg1); 
  return lgp;
}

__global__
void transB(double *vr_rtm, const double* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int order=0, deg=0, j[3]={0,0,0};
  
  // P(m,m)[cos theta]
  
// **** Memory needs to be freed at the end of the function
  double *p_mn_l_0 = (double*) malloc (workLoad * 8);
  double *p_mn_l_1 = (double*) malloc (workLoad * 8);
  double *p_m_l_0 = (double*) malloc (workLoad * 8);
  double *p_m_l_1 = (double*) malloc (workLoad * 8);
  double *p_mp_l_0 = (double*) malloc (workLoad * 8);
  double *p_mp_l_1 = (double*) malloc (workLoad * 8);
  double *dp_m_l_0 = (double*) malloc (workLoad * 8);
  double *dp_m_l_1 = (double*) malloc (workLoad * 8);
 
  double x=1, theta=0;
  double g_sph_rlm_dev[3] = {0,0,0};
// 3 for m-1, m, m+1
  unsigned int idx[3] = {0,0,0}, idx_rtm[3] = {0,0,0};
  double reg1, reg2, reg3;

  double *vr_reg = (double*) malloc (3*sizeof(double)*devConstants.nvector);

  //base case, m=0,1 l=0...t_lvl
  for(int i=0; i<workLoad; i++) {
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    memset(vr_reg, 0, 3*devConstants.nvector);
    // m=0, l=0 && m=0, l=1
    p_m_l_0[i*blockDim.x + id] = 1;
    p_m_l_1[i*blockDim.x + id] = x;
    // m=1, l=1 && m=1, l=2
    p_mp_l_0[i*blockDim.x + id] = calculateLGP_m_eq_l(1);
    p_mp_l_1[i*blockDim.x + id] = calculateLGP_mp1_eq_l(1, x, p_mp_l_0[i*blockDim.x + id]);
    // m=0, l=0 && m=0, l=1
    dp_m_l_0[i*blockDim.x + id] = 0;
    dp_m_l_1[i*blockDim.x + id] = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0[i*blockDim.x + id], theta));

    for(int l=order; l<=devConstants.t_lvl; l++) {
      j[1] = l*(l+1) + 0;
      j[2] = l*(l+1) + 1;

      idx[1] = devConstants.ncomp * (j[1] * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      idx[2] = devConstants.ncomp * (j[2] * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 

      g_sph_rlm_dev[1] = (double) l*(l+1); 
      
      for(int t=1; t<=devConstants.nvector; t++) {
        idx[1] += 3;
        idx[2] += 3;

        vr_reg[t*3 - 3] += sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * p_m_l_0[blockIdx.x*i + id] * g_sph_rlm_dev[1];    
        vr_reg[t*3 - 2] += sp_rlm[idx[1] - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[blockIdx.x*i + id];    
        vr_reg[t*3 - 1] -= sp_rlm[idx[1] - 3] * a_r_1d_rlm_r[blockIdx.x] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[blockIdx.x*i + id];    

      }

      reg1 = nextLGP_m_eq0(l+2, x, p_m_l_0[i*blockDim.x + id], p_m_l_1[i*blockDim.x + id]); 
      p_m_l_0[i*blockDim.x + id] = p_m_l_1[blockDim.x + id];
      p_m_l_1[i*blockDim.x + id] = reg1;

      reg2 = calculateLGP_m_l(1, l+3, theta, p_mp_l_0[blockIdx.x + id], p_mp_l_1[blockIdx.x + id]);
      p_mp_l_0[i*blockDim.x + id] = p_mp_l_1[blockIdx.x + id];
      p_mp_l_1[i*blockDim.x + id] = reg2;

      dp_m_l_0[i*blockDim.x + id] = dp_m_l_1[i*blockDim.x + id];
      dp_m_l_1[i*blockDim.x + id] = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0[i*blockDim.x + id], theta));
        
    }
    
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + ((devConstants.t_lvl + 1)-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*3 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*3 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*3 - 1]; 
    }
  }     
   
  free(p_mn_l_0);
  free(p_mn_l_1);
  free(p_m_l_0);
  free(p_m_l_1);
  free(p_mp_l_0);
  free(p_mp_l_1);
  free(dp_m_l_0);
  free(dp_m_l_1);
  
  // asin_theta_1d_rtm = 1/sin(g_colat_rtm(n_theta)) 
  //Associated Leg Poly for l, m=l
  // ... for l, m=(l-1)
  /*
  for(int k=1; k<=(order-1); k++)
    c_0[1] *= __ddiv_ru((double)2*k-1, (double)2*k);
  c_0[0] = __dmul_ru(c_0[1], __ddiv_ru((double)2*k-1, (double)2*k));

  // 
  for(int i=0; i<workLoad; i++) {
    x = cos(g_colat_rtm[threadIdx.x + i*blockDim.x]); 
    p_0[i] = __dsqrt_rd(__dmul_rd(2, c_0));
    reg1 = p_1[i] =  __dmul_rd(cos(x), __dsqrt_rd(__dmul_rd(2*(2*order+1), c_0)));
      
      
  }
 
  int m=0, l=0;
 
    
  int idx=0, j=0;
  double c_1=c1(order),x=0, c_0=1;
  double reg1=0, reg2=0;
  for(int k=1; k<=order; k++) 
    c_0 *= __ddiv_ru((double)2*k-1, (double)2*k);
  
  
  for(int i=0; i<workLoad; i++) {
    idx = id+i*blockDim.x;
    reg1 = p_0[i] = __dsqrt_rd(__dmul_rd(2, c_0));
    j = l*(l+1) + order;
    x = g_colat[idx];
    for(int k=0; k<order; k++)
      reg1 *= sin(x);
    P_smdt[idx + j*theta] = reg1;
//    lgp += p_0[i] * weights[j];
  }
   */  
  /*vector[id] = lgp;
  __syncthreads();
  int count=0;
  //call convolution kernel
  for(lgp=0; count<blockDim.x; count++) {
    lgp += vector[count];
  }*/

  /*l++;
  for(int i=0; i<workLoad && l <= degree; i++) {
    idx = id+i*blockDim.x;
    x = g_colat[idx];
    reg1 = p_1[i] =  __dmul_rd(cos(x), __dsqrt_rd(__dmul_rd(2*(2*order+1), c_0)));
    j = l*(l+1) + order;
    for(int k=0; k<order; k++)
      reg1 *= sin(x);
    P_smdt[idx + j*theta] = reg1;
    //lgp += p_1[i] * weights[j];
  }
 */ 
/*  vector[id]=lgp;
  __syncthreads();
  //call convolution kernel
  if (l <= degree) {
    for(count=0,lgp=0; count<blockDim.x; count++) {
      lgp += vector[count];
    }
  }
  else
   return;
*/
  /*l++;

  double p_2=0, c_2=0;
  for(; l <= degree; l++) {
    c_1 = c1(l);
    c_2 = c2(order, l);
    for(int i=0; i<workLoad; i++) {
      idx = id+i*blockDim.x;
      x = g_colat[idx];
      p_2 = __ddiv_rd(__dadd_rd(__dmul_rd(c_1, __dmul_rd(cos(x), p_1[i])), __dmul_rd(p_0[i], c_2)), __dsqrt_rd((l*l) - (order*order)));
      p_0[i] = p_1[i];
      p_1[i] = p_2;
      for(int m=0; m<order; m++)
        p_2 *= sin(x);
      j = l*(l+1) + order;
      P_smdt[idx + j*theta] = p_2;
      //lgp+= p_2*weights[j];
    } */
/*
    vector[id]= lgp;
    __syncthreads();
    //call convolution kernel
    for(count=0,lgp=0; count<blockDim.x; count++) {
      lgp += vector[count];
    }
*/
  //}

}

__global__
void transB_m_l_eq0_ver1D(const int mp_rlm, const int jst, const int jed, double *vr_rtm, double *sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *P_smdt, double *dP_smdt, double *g_sph_rlm, double *asin_theta_1d_rtm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int order=0, j=0;
  // P(m,m)[cos theta]
  
// **** Memory needs to be freed at the end of the function
  double p_m_l_0, p_m_l_1;
  double p_mp_l_0, p_mp_l_1;
  double dp_m_l;
 
  double x=1, theta=0;
  unsigned int idx = 0, idx2 = 0, idx_rtm[2] = {0,0};
  double reg1, reg2;

  double *vr_reg = (double*) malloc (5*sizeof(double)*devConstants.nvector);
  double *vr_reg_scalar = (double*) malloc (sizeof(double)*devConstants.nscalar);

  int mn_rlm = devConstants.nidx_rtm[2] - mp_rlm + 1;

  int l=0;
  //base case, m=0,1 l=0...t_lvl
  for(int i=0; i<workLoad; i++) {
    l = 0;
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    for(int rt=0; rt<5*devConstants.nvector;rt++)
      vr_reg[rt]=0;
    for(int rt=0; rt<devConstants.nscalar;rt++)
      vr_reg_scalar[rt]=0;
    // m=0, l=0 && m=0, l=1
    p_m_l_0 = 1;
    #ifdef CUDA_DEBUG
      P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + 0] = p_m_l_0;
    #endif
    p_m_l_1 = x;
    // m=1, l=1 && m=1, l=2
    p_mp_l_0 = calculateLGP_m_eq_l(1);
    p_mp_l_1 = calculateLGP_mp1_eq_l(1, x, p_mp_l_0);
    // m=0, l=0 
    dp_m_l = 0;
    #ifdef CUDA_DEBUG
      dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + 0] = dp_m_l;
    #endif
    dp_m_l = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0, theta));

    for(int j_rlm=jst; j_rlm<=jed; j_rlm++, l++) {
      idx = devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      idx2 = 3*devConstants.nvector + devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      for(int t=1; t<=devConstants.nvector; t++) {
        idx += 3;
        reg2 = __dmul_rd(__dmul_rd(-1 * p_m_l_0, (double) order), asin_theta_1d_rtm[id + i*blockDim.x]);         
        vr_reg[t*5 - 5] += sp_rlm[idx - 1] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        vr_reg[t*5 - 4] += sp_rlm[idx - 2] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        vr_reg[t*5 - 3] += sp_rlm[idx - 3] * __dmul_rd(a_r_1d_rlm_r[blockIdx.x], a_r_1d_rlm_r[blockIdx.x]) * p_m_l_0 * g_sph_rlm[j_rlm-1];    
        vr_reg[t*5 - 2] += sp_rlm[idx - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;    
        vr_reg[t*5 - 1] -= sp_rlm[idx - 1] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;
      }
      for(int t=1; t<=devConstants.nscalar; t++) {
        idx2 += 1;
        vr_reg_scalar[t-1] += sp_rlm[idx2 - 1] * p_m_l_0;
      }

      //m=0, l=l+2
      reg1 = nextLGP_m_eq0(l+2, x, p_m_l_0, p_m_l_1); 
      //P_smdt is set to be nTheta x nJ
      //m=0, l=l+1
      j = (l+1)*(l+2) + 0;
      p_m_l_0 = p_m_l_1;
      #if defined(CUDA_DEBUG)
      if(l<=devConstants.t_lvl)
        P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = p_m_l_0; 
      #endif
      p_m_l_1 = reg1;

      //m=0, l=l+1
      j = (l+1)*(l+2) + 0;
      dp_m_l = nextDp_m_eq_0(l+1, scaleBySine(1, p_mp_l_0, theta));
      #ifdef CUDA_DEBUG
      if(l<=devConstants.t_lvl)
        dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = dp_m_l;
      #endif

      //m=1, l=l+3
      reg2 = calculateLGP_m_l(1, l+3, theta, p_mp_l_0, p_mp_l_1);
      p_mp_l_0 = p_mp_l_1;
      p_mp_l_1 = reg2;
        
    }
    
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    idx_rtm[1] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mn_rlm-1) * devConstants.istep_rtm[2]); 
    idx2 = devConstants.nvector*3 + devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      idx_rtm[1] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*5 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*5 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*5 - 1]; 
      vr_rtm[idx_rtm[1] - 1 - 1] += vr_reg[t*5 - 5];
      vr_rtm[idx_rtm[1] - 1] += vr_reg[t*5 - 4];  
    }
    for(int t=1; t<=devConstants.nscalar; t++) {
      idx2 += 1;
      vr_rtm[idx2 - 1] = vr_reg_scalar[t-1];
    } 
  }     
   
  free(vr_reg);
  free(vr_reg_scalar);
}

__global__
void transB_m_l_eq1_ver1D(const int mp_rlm, const int jst, const int jed, int order, int degree, double *vr_rtm, double *sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *P_smdt, double *dP_smdt, double *g_sph_rlm, double *asin_theta_1d_rtm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int deg=0, j=0;
  // P(m,m)[cos theta]
  
  double p_mn_l_0, p_mn_l_1;
  double p_m_l_0, p_m_l_1;
  double p_mp_l_0, p_mp_l_1;
  double dp_m_l;
 
  double x=1, theta=0;
// 3 for m-1, m, m+1
  unsigned int idx[3] = {0,0,0}, idx_rtm[3] = {0,0,0};
  double reg1, reg2, reg3;

  double *vr_reg = (double*) malloc (5*sizeof(double)*devConstants.nvector);
  double *vr_reg_scalar = (double*) malloc (sizeof(double)*devConstants.nscalar);

  int mn_rlm = devConstants.nidx_rtm[2] - mp_rlm + 1;

  for(int i=0; i<workLoad; i++) {
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    // deg = 1 = order
    deg = abs(degree);
    for(int rt=0; rt<5*devConstants.nvector;rt++)
      vr_reg[rt]=0;
    for(int rt=0; rt<devConstants.nscalar;rt++)
      vr_reg_scalar[rt]=0;
    
    //m=0, l=0
    p_mn_l_0 = 1;
    //m=0, l=1
    p_mn_l_1 = x;

    //m=l=1
    p_m_l_0 = calculateLGP_m_eq_l(1); 
    #ifdef CUDA_DEBUG
      j = degree * (degree+1) + order;
      P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(1, p_m_l_0, theta);
    #endif
    //m = 1, l=2
    p_m_l_1 = calculateLGP_mp1_eq_l(1, x, p_m_l_0);

   //m = l = 2
    p_mp_l_0 = calculateLGP_m_eq_l(2);
   //m=2, l=3
    p_mp_l_1 = calculateLGP_mp1_eq_l(2, x, p_mp_l_0);

    //m=l=1
    dp_m_l = p_mn_l_1;
    #ifdef CUDA_DEBUG
      j = degree * (degree+1) + order;
      dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = dp_m_l; 
    #endif

    for(int j_rlm=jst, deg=degree; j_rlm<=jed; j_rlm++, deg++) {
      idx[1] = devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      idx[0] = devConstants.nvector*3 + devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      for(int t=1; t<=devConstants.nvector; t++) {
        idx[1] += 3;
        reg2 = __dmul_rd(__dmul_rd(-1 * scaleBySine(order, p_m_l_0, theta), (double) order), asin_theta_1d_rtm[id + i*blockDim.x]);         
        vr_reg[t*5 - 5] += sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        vr_reg[t*5 - 4] += sp_rlm[idx[1] - 2] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        vr_reg[t*5 - 3] += sp_rlm[idx[1] - 3] * __dmul_rd(a_r_1d_rlm_r[blockIdx.x], a_r_1d_rlm_r[blockIdx.x]) * scaleBySine(order, p_m_l_0, theta) * g_sph_rlm[j_rlm-1];    
        vr_reg[t*5 - 2] += sp_rlm[idx[1] - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;    
        vr_reg[t*5 - 1] -= sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;    
      }
      for(int t=1; t<=devConstants.nscalar; t++) {
        idx[0] += 1;
        vr_reg_scalar[t-1] += sp_rlm[idx[0] - 1] * scaleBySine(order, p_m_l_0, theta);
      }

      // Initially l and m are set to 1
      //m=0, l = l+1 
      reg1 = nextLGP_m_eq0(deg+1, x, p_mn_l_0, p_mn_l_1); 
      p_mn_l_0 = p_mn_l_1;
      p_mn_l_1 = reg1;

      //m=1, l = l+2
      reg2 = calculateLGP_m_l(order, deg+2, theta, p_m_l_0, p_m_l_1);
      p_m_l_0 = p_m_l_1;
      //setting debug data for m=1, l = l+1
      #ifdef CUDA_DEBUG
        if(deg <= devConstants.t_lvl) {
          j = (deg+1)*(deg+2) + order;
          P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order, p_m_l_0, theta);
        }
      #endif
      p_m_l_1 = reg2;
 
      //m=1, l = l+1
      dp_m_l = nextDp_m_eq_1(deg+1, p_mn_l_1, scaleBySine(order+1, p_mp_l_0, theta));
       
      #ifdef CUDA_DEBUG
        if(deg<=devConstants.t_lvl) {
          j = (deg+1)*(deg+2) + order;
          dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = dp_m_l; 
        }
      #endif

      //m=order+1, l=l+3
      reg3 = calculateLGP_m_l(order+1, deg+3, theta, p_mp_l_0, p_mp_l_1);  
      p_mp_l_0 = p_mp_l_1;
      //setting debug info for m=order+1, l=l+2
      //j = (deg+2)*(deg+3) + order+1;
 //     P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order+1, p_mp_l_0, theta);
      p_mp_l_1 = reg3;
        
    }
    
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    idx_rtm[1] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mn_rlm-1) * devConstants.istep_rtm[2]); 
    idx_rtm[2] = 3*devConstants.nvector + devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      idx_rtm[1] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*5 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*5 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*5 - 1]; 
      vr_rtm[idx_rtm[1] - 1 - 1] += vr_reg[t*5 - 5];
      vr_rtm[idx_rtm[1] - 1] += vr_reg[t*5 - 4];  
    }
    for(int t=1; t<=devConstants.nscalar; t++) {
      idx_rtm[2] += 1;
      vr_rtm[idx_rtm[2] - 1] = vr_reg_scalar[t-1];
    }
  }     
   
  free(vr_reg);
  free(vr_reg_scalar);
}

__global__
void transB_m_l_ver1D(const int mp_rlm, const int jst, const int jed, int order, int degree, double *vr_rtm, double *sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *P_smdt, double *dP_smdt, double *g_sph_rlm, double *asin_theta_1d_rtm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int deg=0, j=0;
  // P(m,m)[cos theta]
  
  double p_mn_l_0, p_mn_l_1;
  double p_m_l_0, p_m_l_1;
  double p_mp_l_0, p_mp_l_1;
  double dp_m_l;
 
  double x=1, theta=0;
// 3 for m-1, m, m+1
  unsigned int idx[3] = {0,0,0}, idx_rtm[3] = {0,0,0};
  double reg1, reg2, reg3;

  double *vr_reg = (double*) malloc (sizeof(double)*5*devConstants.nvector);
  double *vr_reg_scalar = (double*) malloc (sizeof(double)*devConstants.nscalar);

  double mn_rlm = devConstants.nidx_rtm[2] - mp_rlm + 1;

  for(int i=0; i<workLoad; i++) {
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    deg = degree;

    for(int rt=0; rt<devConstants.nvector*5;rt++)
      vr_reg[rt]=0;
    for(int rt=0; rt<devConstants.nscalar;rt++)
      vr_reg_scalar[rt]=0;
   
    // m-1,l-1
    p_mn_l_0 = calculateLGP_m_eq_l(abs(order)-1);
    //P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order-1, p_mn_l_0, theta);
    // m-1,l
    p_mn_l_1 = calculateLGP_mp1_eq_l(abs(order)-1, x, p_mn_l_0);

    // m,l
    p_m_l_0 = calculateLGP_m_eq_l(order); 
    
    #ifdef CUDA_DEBUG
    j = degree*(degree+1) + order;
    P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order, p_m_l_0, theta);
    #endif

    // m,l+1
    p_m_l_1 = calculateLGP_mp1_eq_l(order, x, p_m_l_0);

    // m+1,l+1 
    p_mp_l_0 = calculateLGP_m_eq_l(abs(order)+1);
    // m+1,l+2
    p_mp_l_1 = calculateLGP_mp1_eq_l(abs(order)+1, x, p_mp_l_0);

    // m,l
    dp_m_l = __dmul_rd(0.5, __dmul_rd(__dsqrt_rd(2*abs(order)), scaleBySine(abs(order)-1, p_mn_l_1, theta)));
    #ifdef CUDA_DEBUG
      j = degree*(degree+1) + order;
      dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = dp_m_l; 
    #endif

    for(int j_rlm=jst, deg=degree; j_rlm<=jed; j_rlm++, deg++) {
      idx[1] = devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      idx[0] = 3*devConstants.nvector + devConstants.ncomp * ((j_rlm-1) * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      for(int t=1; t<=devConstants.nvector; t++) {
        idx[1] += 3;
        reg1 = scaleBySine(order, p_m_l_0, theta);
        reg1 *= g_sph_rlm[j_rlm-1];
        reg2 = __dmul_rd(__dmul_rd(-1 * scaleBySine(order, p_m_l_0, theta), (double) order), asin_theta_1d_rtm[id + i*blockDim.x]);         
        vr_reg[t*5 - 5] += sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        vr_reg[t*5 - 4] += sp_rlm[idx[1] - 2] * a_r_1d_rlm_r[blockIdx.x] * reg2;
        //vr_reg[t*3 - 3] += sp_rlm[idx[1] - 3] * __dmul_rd(a_r_1d_rlm_r[blockIdx.x], a_r_1d_rlm_r[blockIdx.x]) * scaleBySine(order, p_m_l_0, theta) * g_sph_rlm[j_rlm-1];    
        vr_reg[t*5 - 3] += sp_rlm[idx[1] - 3] * __dmul_rd(a_r_1d_rlm_r[blockIdx.x], a_r_1d_rlm_r[blockIdx.x]) * reg1;    
        vr_reg[t*5 - 2] += sp_rlm[idx[1] - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;    
        vr_reg[t*5 - 1] -= sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l;    
      }
      
      for(int t=1; t<=devConstants.nscalar; t++) {
        idx[0] += 1;
        vr_reg_scalar[t-1] += sp_rlm[idx[0] - 1] * scaleBySine(order, p_m_l_0, theta);
      } 

      // m-1, l+1 
      reg1 = calculateLGP_m_l(abs(order)-1, deg+1, theta, p_mn_l_0, p_mn_l_1); 
      p_mn_l_0 = p_mn_l_1;
      p_mn_l_1 = reg1;

      // m, l+2
      reg2 = calculateLGP_m_l(order, deg+2, theta, p_m_l_0, p_m_l_1);
      p_m_l_0 = p_m_l_1;
      // p_m_l_0, m, l+1
      #ifdef CUDA_DEBUG
        if(deg<=devConstants.t_lvl) 
          j = (deg+1)*(deg+2) + order;
          P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order, p_m_l_0, theta);
      #endif
      p_m_l_1 = reg2;
 
      //m, l+1
      dp_m_l = nextDp_m_l(order, deg+1, scaleBySine(abs(order)-1, p_mn_l_1, theta), scaleBySine(abs(order)+1, p_mp_l_0, theta));
     
      #ifdef CUDA_DEBUG
        if(deg<=devConstants.t_lvl)
          j = (deg+1)*(deg+2) + order;
          dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = dp_m_l; 
      #endif
      //m+1, l+3
      reg3 = calculateLGP_m_l(abs(order)+1, deg+3, theta, p_mp_l_0, p_mp_l_1);  
      p_mp_l_0 = p_mp_l_1;
      // p_mp_1_0, m+1, l+2
      //j = (deg+2)*(deg+3) + order+1;
      //P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j] = scaleBySine(order+1, p_mp_l_0, theta);
      p_mp_l_1 = reg3;
        
    }
    // mp_rlm 
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    // mn_rlm
    idx_rtm[1] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mn_rlm-1) * devConstants.istep_rtm[2]); 
    idx_rtm[2] = 3*devConstants.nvector + devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + (mp_rlm-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      idx_rtm[1] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*5 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*5 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*5 - 1]; 
      vr_rtm[idx_rtm[1] - 1 - 1] += vr_reg[t*5 - 5]; 
      vr_rtm[idx_rtm[1] - 1] += vr_reg[t*5 - 4]; 
    }
    for(int t=1; t<=devConstants.nscalar; t++) { 
      idx_rtm[2] += 1;
      vr_rtm[idx_rtm[2] - 1] = vr_reg_scalar[t-1];
    }
  }     
   
  free(vr_reg);
  free(vr_reg_scalar);
}

void legendre_b_trans_vector_cuda_(int *ncomp, int *nvector, int *nscalar) {
  
//  static int nShells = *ked - *kst + 1;
  static int nShells = constants.nidx_rtm[0];
  static int nTheta = constants.nidx_rtm[1];

  constants.ncomp = *ncomp; 
  constants.nscalar = *nscalar;
  constants.nvector = *nvector;

  initDevConstVariables();
   
  dim3 grid(nShells, 1);
  dim3 block(16,1,1);  
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm, 3 = a_r_1d_rlm_r

  int jst, jed, m, l;
  for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++) {
    jst = h_debug.lstack_rlm[mp_rlm-1] + 1;
    jed = h_debug.lstack_rlm[mp_rlm];
    m = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + jst - 1]; 
    l = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1]*1 + jst - 1]; 

    if(l==0) {
        transB_m_l_eq0_ver1D<<<grid, block, 0, streams[l%32]>>> (mp_rlm, jst, jed, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, d_debug.P_smdt, d_debug.dP_smdt, deviceInput.g_sph_rlm, deviceInput.asin_theta_1d_rtm);
    }
    else if (l==1) {
        transB_m_l_eq1_ver1D<<<grid, block, 0, streams[l%32]>>> (mp_rlm, jst, jed, m, l, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, d_debug.P_smdt, d_debug.dP_smdt, deviceInput.g_sph_rlm, deviceInput.asin_theta_1d_rtm);
    }
    else { 
        transB_m_l_ver1D<<<grid, block, 0, streams[l%32]>>> (mp_rlm, jst, jed, m, l, deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, d_debug.P_smdt, d_debug.dP_smdt, deviceInput.g_sph_rlm, deviceInput.asin_theta_1d_rtm);
    } 
//    cudaErrorCheck(hipDeviceSynchronize());
//    cpy_dev2host_4_debug_();
  } 

#ifdef CUDA_TIMINGS
  cudaErrorCheck(hipDeviceSynchronize());
#endif
}


