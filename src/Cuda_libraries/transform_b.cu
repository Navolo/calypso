#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>

__global__
void transB(double *vr_rtm, double *sp_rlm, double *g_sph_rlm) {

  int nidx_theta = blockIdx.x;
  
}

void transform_b_(int *ncomp, int *nvector, int *nscalar) {
  dim3 grid(nidx_rtm[1]);
  dim3 block(nidx_rtm[0]);  
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm
  transB<<<grid, block>>> (deviceInput.argv[0], deviceInput.argv[1], deviceInput.argv[2]);
}
