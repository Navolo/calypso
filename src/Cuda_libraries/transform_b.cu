#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <sstream>

__device__
double nextLGP_m_eq0(int l, double x, double p_0, double p_1) {
// x = cos(theta)
  return __dadd_rd(__dmul_rd(__dmul_rd(p_1, __ddiv_rd(2*l-1, l)),x), __dmul_rd(p_0, __ddiv_rd(l-1, l))*-1); 
}

__device__
double nextDp_m_eq_0(int l, double lgp_mp) {
   return __dmul_rd(-1*lgp_mp, __dsqrt_rd(l*(l+1)/2)); 
}

__device__
double calculateLGP_m_eq_l(int mode) {
  double lgp=1;
  for(int k=1; k<=mode; k++) {
    lgp *= __ddiv_ru((double)2*k-1, (double)2*k);
  }
  
  return __dsqrt_rd(__dmul_rd(2, lgp));
}

__device__ 
double calculateLGP_mp1_eq_l(int mode, double x, double lgp_m_eq_l) {
  return __dmul_rd(__dmul_rd(lgp_m_eq_l, __dsqrt_rd(2*mode+1)),x); 
}

__device__
double calculateLGP_m_l(int m, int degree, double theta, double lgp_0, double lgp_1) {
  return  __ddiv_rd(__dadd_rd(__dmul_rd(2*degree-1, __dmul_rd(cos(theta), lgp_1)), __dmul_rd(-1 * lgp_0, __dsqrt_rd((degree-1)*(degree-1) - m*m))), __dsqrt_rd((degree*degree) - (m*m)));
}

__device__
double scaleBySine(int m, double lgp, double theta) {
  double reg1 = sin(theta);
  for(int r=0; r<m; r++)
    lgp = __dmul_rd(lgp,reg1); 
  return lgp;
}

__global__
void transB(double *vr_rtm, const double* __restrict__ sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int order=0, deg=0, j[3]={0,0,0};
  
  // P(m,m)[cos theta]
  
// **** Memory needs to be freed at the end of the function
  double *p_mn_l_0 = (double*) malloc (workLoad * 8);
  double *p_mn_l_1 = (double*) malloc (workLoad * 8);
  double *p_m_l_0 = (double*) malloc (workLoad * 8);
  double *p_m_l_1 = (double*) malloc (workLoad * 8);
  double *p_mp_l_0 = (double*) malloc (workLoad * 8);
  double *p_mp_l_1 = (double*) malloc (workLoad * 8);
  double *dp_m_l_0 = (double*) malloc (workLoad * 8);
  double *dp_m_l_1 = (double*) malloc (workLoad * 8);
 
  double x=1, theta=0;
  double g_sph_rlm_dev[3] = {0,0,0};
// 3 for m-1, m, m+1
  unsigned int idx[3] = {0,0,0}, idx_rtm[3] = {0,0,0};
  double reg1, reg2, reg3;

  double *vr_reg = (double*) malloc (3*sizeof(double)*devConstants.nvector);

  //base case, m=0,1 l=0...t_lvl
  for(int i=0; i<workLoad; i++) {
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    memset(vr_reg, 0, 3*devConstants.nvector);
    // m=0, l=0 && m=0, l=1
    p_m_l_0[i*blockDim.x + id] = 1;
    p_m_l_1[i*blockDim.x + id] = x;
    // m=1, l=1 && m=1, l=2
    p_mp_l_0[i*blockDim.x + id] = calculateLGP_m_eq_l(1);
    p_mp_l_1[i*blockDim.x + id] = calculateLGP_mp1_eq_l(1, x, p_mp_l_0[i*blockDim.x + id]);
    // m=0, l=0 && m=0, l=1
    dp_m_l_0[i*blockDim.x + id] = 0;
    dp_m_l_1[i*blockDim.x + id] = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0[i*blockDim.x + id], theta));

    for(int l=order; l<=devConstants.t_lvl; l++) {
      j[1] = l*(l+1) + 0;
      j[2] = l*(l+1) + 1;

      idx[1] = devConstants.ncomp * (j[1] * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      idx[2] = devConstants.ncomp * (j[2] * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 

      g_sph_rlm_dev[1] = (double) l*(l+1); 
      
      for(int t=1; t<=devConstants.nvector; t++) {
        idx[1] += 3;
        idx[2] += 3;

        vr_reg[t*3 - 3] += sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * p_m_l_0[blockIdx.x*i + id] * g_sph_rlm_dev[1];    
        vr_reg[t*3 - 2] += sp_rlm[idx[1] - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[blockIdx.x*i + id];    
        vr_reg[t*3 - 1] -= sp_rlm[idx[1] - 3] * a_r_1d_rlm_r[blockIdx.x] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[blockIdx.x*i + id];    

      }

      reg1 = nextLGP_m_eq0(l+2, x, p_m_l_0[i*blockDim.x + id], p_m_l_1[i*blockDim.x + id]); 
      p_m_l_0[i*blockDim.x + id] = p_m_l_1[blockDim.x + id];
      p_m_l_1[i*blockDim.x + id] = reg1;

      reg2 = calculateLGP_m_l(1, l+3, theta, p_mp_l_0[blockIdx.x + id], p_mp_l_1[blockIdx.x + id]);
      p_mp_l_0[i*blockDim.x + id] = p_mp_l_1[blockIdx.x + id];
      p_mp_l_1[i*blockDim.x + id] = reg2;

      dp_m_l_0[i*blockDim.x + id] = dp_m_l_1[i*blockDim.x + id];
      dp_m_l_1[i*blockDim.x + id] = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0[i*blockDim.x + id], theta));
        
    }
    
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + ((devConstants.t_lvl + 1)-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*3 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*3 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*3 - 1]; 
    }
  }     
   
  free(p_mn_l_0);
  free(p_mn_l_1);
  free(p_m_l_0);
  free(p_m_l_1);
  free(p_mp_l_0);
  free(p_mp_l_1);
  free(dp_m_l_0);
  free(dp_m_l_1);
  
  // asin_theta_1d_rtm = 1/sin(g_colat_rtm(n_theta)) 
  //Associated Leg Poly for l, m=l
  // ... for l, m=(l-1)
  /*
  for(int k=1; k<=(order-1); k++)
    c_0[1] *= __ddiv_ru((double)2*k-1, (double)2*k);
  c_0[0] = __dmul_ru(c_0[1], __ddiv_ru((double)2*k-1, (double)2*k));

  // 
  for(int i=0; i<workLoad; i++) {
    x = cos(g_colat_rtm[threadIdx.x + i*blockDim.x]); 
    p_0[i] = __dsqrt_rd(__dmul_rd(2, c_0));
    reg1 = p_1[i] =  __dmul_rd(cos(x), __dsqrt_rd(__dmul_rd(2*(2*order+1), c_0)));
      
      
  }
 
  int m=0, l=0;
 
    
  int idx=0, j=0;
  double c_1=c1(order),x=0, c_0=1;
  double reg1=0, reg2=0;
  for(int k=1; k<=order; k++) 
    c_0 *= __ddiv_ru((double)2*k-1, (double)2*k);
  
  
  for(int i=0; i<workLoad; i++) {
    idx = id+i*blockDim.x;
    reg1 = p_0[i] = __dsqrt_rd(__dmul_rd(2, c_0));
    j = l*(l+1) + order;
    x = g_colat[idx];
    for(int k=0; k<order; k++)
      reg1 *= sin(x);
    P_smdt[idx + j*theta] = reg1;
//    lgp += p_0[i] * weights[j];
  }
   */  
  /*vector[id] = lgp;
  __syncthreads();
  int count=0;
  //call convolution kernel
  for(lgp=0; count<blockDim.x; count++) {
    lgp += vector[count];
  }*/

  /*l++;
  for(int i=0; i<workLoad && l <= degree; i++) {
    idx = id+i*blockDim.x;
    x = g_colat[idx];
    reg1 = p_1[i] =  __dmul_rd(cos(x), __dsqrt_rd(__dmul_rd(2*(2*order+1), c_0)));
    j = l*(l+1) + order;
    for(int k=0; k<order; k++)
      reg1 *= sin(x);
    P_smdt[idx + j*theta] = reg1;
    //lgp += p_1[i] * weights[j];
  }
 */ 
/*  vector[id]=lgp;
  __syncthreads();
  //call convolution kernel
  if (l <= degree) {
    for(count=0,lgp=0; count<blockDim.x; count++) {
      lgp += vector[count];
    }
  }
  else
   return;
*/
  /*l++;

  double p_2=0, c_2=0;
  for(; l <= degree; l++) {
    c_1 = c1(l);
    c_2 = c2(order, l);
    for(int i=0; i<workLoad; i++) {
      idx = id+i*blockDim.x;
      x = g_colat[idx];
      p_2 = __ddiv_rd(__dadd_rd(__dmul_rd(c_1, __dmul_rd(cos(x), p_1[i])), __dmul_rd(p_0[i], c_2)), __dsqrt_rd((l*l) - (order*order)));
      p_0[i] = p_1[i];
      p_1[i] = p_2;
      for(int m=0; m<order; m++)
        p_2 *= sin(x);
      j = l*(l+1) + order;
      P_smdt[idx + j*theta] = p_2;
      //lgp+= p_2*weights[j];
    } */
/*
    vector[id]= lgp;
    __syncthreads();
    //call convolution kernel
    for(count=0,lgp=0; count<blockDim.x; count++) {
      lgp += vector[count];
    }
*/
  //}

}

__global__
void transB(double *vr_rtm, double *sp_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *P_smdt, double *dP_smdt, double *g_sph_rlm, int *lstack_rlm) {
  unsigned int id = threadIdx.x;
  // The work is parallelized over theta within a grid
  int nTheta = devConstants.nidx_rtm[1];
  unsigned int workLoad = nTheta/(blockDim.x);
  if( nTheta%blockDim.x > (threadIdx.x)) 
    workLoad++;

  int order=0, deg=0, j[3]={0,0,0};
  int jst=0, jed=0; 
  // P(m,m)[cos theta]
  
// **** Memory needs to be freed at the end of the function
  double *p_mn_l_0 = (double*) malloc (workLoad * 8);
  double *p_mn_l_1 = (double*) malloc (workLoad * 8);
  double *p_m_l_0 = (double*) malloc (workLoad * 8);
  double *p_m_l_1 = (double*) malloc (workLoad * 8);
  double *p_mp_l_0 = (double*) malloc (workLoad * 8);
  double *p_mp_l_1 = (double*) malloc (workLoad * 8);
  double *dp_m_l_0 = (double*) malloc (workLoad * 8);
  double *dp_m_l_1 = (double*) malloc (workLoad * 8);
 
  double x=1, theta=0;
// 3 for m-1, m, m+1
  unsigned int idx[3] = {0,0,0}, idx_rtm[3] = {0,0,0};
  double reg1, reg2, reg3;

  double *vr_reg = (double*) malloc (3*sizeof(double)*devConstants.nvector);

  int l;
  //base case, m=0,1 l=0...t_lvl
  for(int i=0; i<workLoad; i++) {
    theta = g_colat_rtm[threadIdx.x + i*blockDim.x];
    x = cos(theta);
    for(int rt=0; rt<3*devConstants.nvector;rt++)
      vr_reg[rt]=0;
    // m=0, l=0 && m=0, l=1
    P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + 0] = p_m_l_0[i] = 1;
    p_m_l_1[i] = x;
    // m=1, l=1 && m=1, l=2
    p_mp_l_0[i] = calculateLGP_m_eq_l(1);
    P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + 3] = scaleBySine(1, p_mp_l_0[i], theta);
    p_mp_l_1[i*blockDim.x + id] = calculateLGP_mp1_eq_l(1, x, p_mp_l_0[i]);
    // m=0, l=0 && m=0, l=1
    dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + 0] = dp_m_l_0[i] = 0;
    dp_m_l_1[i] = nextDp_m_eq_0(1, scaleBySine(1, p_mp_l_0[i], theta));

    jst = lstack_rlm[devConstants.t_lvl] + 1;
    jed = lstack_rlm[devConstants.t_lvl+1];
    
    for(int j_rlm=jst, l=order; j_rlm<=jed; j_rlm++, l++) {
      idx[1] = devConstants.ncomp * (j_rlm * devConstants.istep_rlm[1] + blockIdx.x * devConstants.istep_rlm[0]); 
      for(int t=1; t<=devConstants.nvector; t++) {
        idx[1] += 3;
        vr_reg[t*3 - 3] += sp_rlm[idx[1] - 1] * a_r_1d_rlm_r[blockIdx.x] * p_m_l_0[i] * g_sph_rlm[j_rlm-1];    
        vr_reg[t*3 - 2] += sp_rlm[idx[1] - 2]  * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[i];    
        vr_reg[t*3 - 1] -= sp_rlm[idx[1] - 3] * a_r_1d_rlm_r[blockIdx.x] * a_r_1d_rlm_r[blockIdx.x] * dp_m_l_0[i];    
      }

      reg1 = nextLGP_m_eq0(l+2, x, p_m_l_0[i], p_m_l_1[i]); 
      j[0] = (l+1)*(l+2) + 0;
      //P_smdt is set to be nTheta x nJ
      P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j[0]] = p_m_l_0[i] = p_m_l_1[i];
      p_m_l_1[i*blockDim.x + id] = reg1;

      reg2 = calculateLGP_m_l(1, l+3, theta, p_mp_l_0[i], p_mp_l_1[i]);
      j[0] = (l+2)*(l+3) + 1;
      p_mp_l_0[i] = p_mp_l_1[i];
      P_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j[0]] = scaleBySine(1, p_mp_l_0[i], theta); 
      p_mp_l_1[i] = reg2;
   
      j[0] = (l+1)*(l+2) + 0;
      dP_smdt[(i*blockDim.x + id)*devConstants.nidx_rlm[1] + j[0]] = dp_m_l_0[i] = dp_m_l_1[i];
      dp_m_l_1[i] = nextDp_m_eq_0(l+2, scaleBySine(1, p_mp_l_0[i], theta));
        
    }
    
    idx_rtm[0] = devConstants.ncomp * ((threadIdx.x + i*blockDim.x) * devConstants.istep_rtm[1] + blockIdx.x*devConstants.istep_rtm[0] + ((devConstants.t_lvl + 1)-1) * devConstants.istep_rtm[2]); 
    for(int t=1; t<=devConstants.nvector; t++) {
      idx_rtm[0] += 3;
      vr_rtm[idx_rtm[0] - 2 - 1]  += vr_reg[t*3 - 3]; 
      vr_rtm[idx_rtm[0] - 1 - 1]  += vr_reg[t*3 - 2]; 
      vr_rtm[idx_rtm[0] - 1]  += vr_reg[t*3 - 1]; 
    }
  }     
   
  free(p_mn_l_0);
  free(p_mn_l_1);
  free(p_m_l_0);
  free(p_m_l_1);
  free(p_mp_l_0);
  free(p_mp_l_1);
  free(dp_m_l_0);
  free(dp_m_l_1);
  free(vr_reg);
}

void transform_b_(int *ncomp, int *nvector, int *nscalar, double *vr_rtm) {
  
//  static int nShells = *ked - *kst + 1;
  static int nShells = constants.nidx_rtm[0];
  static int nTheta = constants.nidx_rtm[1];

  constants.ncomp = *ncomp; 
  constants.nscalar = *nscalar;
  constants.nvector = *nvector;

  initDevConstVariables();
   
  dim3 grid(nShells, 1);
  //dim3 grid(1, 1);
  dim3 block(1,1,1);  
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm, 3 = a_r_1d_rlm_r

  double iTime, fTime;

#ifdef CUDA_DEBUG  
  countBT++;
  std::ostringstream cvrt;
  cvrt.clear();
  cvrt << countBT;
  std::string fileName = "backwardSHT_" + cvrt.str() + ".dat";      
#endif

  iTime = MPI_Wtime();
#ifdef CUDA_DEBUG  
  transB<<<grid, block>>> (deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm, d_debug.P_smdt, d_debug.dP_smdt, deviceInput.g_sph_rlm, deviceInput.lstack_rlm);
  cudaErrorCheck(hipDeviceSynchronize());
#else
  transB<<<grid, block>>> (deviceInput.vr_rtm, deviceInput.sp_rlm, deviceInput.a_r_1d_rlm_r, deviceInput.g_colat_rtm)
  cudaErrorCheck(hipDeviceSynchronize());
#endif 
  fTime = MPI_Wtime();
 
  clockD << "Backward Transform Time for iteration # " << countBT << " took " << fTime-iTime << std::endl;

  iTime = MPI_Wtime();
  set_physical_data_(vr_rtm); 
  cudaErrorCheck(hipDeviceSynchronize());
  fTime = MPI_Wtime();
  clockD << "Time to copy results of backward SHT from dev to host is " << fTime-iTime << std::endl;

#ifdef CUDA_DEBUG
  cpyDev2Host(&d_debug, &h_debug); 
  cudaErrorCheck(hipDeviceSynchronize());
  h_debug.vr_rtm = vr_rtm;
  writeDebugData2File(&h_debug, fileName);
#endif
}

