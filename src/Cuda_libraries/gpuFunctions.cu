#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>

Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;

double countFT=0, countBT=0;

hipStream_t streams[32];
__constant__ Geometry_c devConstants;

//File Streams
std::ofstream clockD;

void initgpu_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *ncomp, int *trunc_lvl) {

  std::string fileName = "GPUTimings.dat";
  clockD.open(fileName.c_str(), std::ofstream::out);
  
  double t_0, t_1;

  for(int i=0; i<3; i++) { 
    constants.nidx_rtm[i] = nidx_rtm[i];
    constants.istep_rtm[i] = istep_rtm[i];
  }

  for(int i=0; i<2; i++) {
    constants.nidx_rlm[i] = nidx_rlm[i];
    constants.istep_rlm[i] = istep_rlm[i];
  }

  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.ncomp = *ncomp;
  constants.t_lvl = *trunc_lvl; 

  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    hipProfilerStart();
    allocHostDebug(&h_debug);
    allocDevDebug(&d_debug);
  #endif

  allocMemOnGPU();

  for(unsigned int i=0; i<32; i++)       
    cudaErrorCheck(hipStreamCreate(&streams[i]));

  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferEqual));

  #if defined(CUDA_TIMINGS)
    cudaErrorCheck(hipDeviceSynchronize());
  #endif
}

void setptrs_(int *idx_gl_1d_rlm_j) {
  //Necessary to filter harmonic modes across MPI nodes.
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
}

/*void setptrs_(int *idx_gl_1d_rlm_j, double *P_smdt, double *dP_smdt) {
  h_debug.idx_gl_1d_rlm_j = idx_gl_1d_rlm_j;
  //h_debug.P_smdt = P_smdt;
  //h_debug.dP_smdt = dP_smdt;
}*/

void allocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  int ncomp = constants.ncomp;
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_colat_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.asin_theta_1d_rtm), constants.nidx_rtm[1]*sizeof(double))); 
//  cudaErrorCheck(hipMalloc((void**)&(deviceInput.lstack_rlm), (constants.nidx_rtm[2]+1)*sizeof(int))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rlm[1]*sizeof(double))); 
   
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
}

void memcpy_h2d_(int *lstack_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *g_sph_rlm, double *asin_theta_1d_rtm) {
    h_debug.lstack_rlm = lstack_rlm;
 #ifdef CUDA_DEBUG 
    h_debug.g_colat_rtm = g_colat_rtm;
    h_debug.g_sph_rlm = g_sph_rlm;
#endif

  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.asin_theta_1d_rtm, asin_theta_1d_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
//  cudaErrorCheck(hipMemcpy(deviceInput.lstack_rlm, lstack_rlm, (constants.nidx_rtm[2]+1)*sizeof(int), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  #ifdef CUDA_TIMINGS
    cudaErrorCheck(hipDeviceSynchronize()); 
  #endif
}

void cpy_schmidt_2_gpu_(double *P_jl, double *dP_jl) {
  #ifdef CUDA_STATIC
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.P_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.dP_jl), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMemcpy(P_jl, deviceInput.P_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMemcpy(dP_jl, deviceInput.dP_jl, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  #endif
}
 
void allocHostDebug(Debug* h_data) {
  h_debug.P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
  h_debug.dP_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
  #if defined(CUDA_DEBUG)
  h_debug.vr_rtm = (double*) malloc (sizeof(double)*constants.nnod_rtm*constants.ncomp);
  #endif
//  h_data->g_sph_rlm = (double*) malloc (sizeof(double)*constants.nidx_rlm[1]);
}
 
void allocDevDebug(Debug* d_data) {
  cudaErrorCheck(hipMalloc((void**)&(d_data->P_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMemset(d_data->P_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(d_data->dP_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMemset(d_data->dP_smdt, -1, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
//  cudaErrorCheck(hipMalloc((void**)&(d_data->g_sph_rlm), sizeof(double)*constants.nidx_rlm[1]));
}

void cpy_dev2host_4_debug_() {
  cudaErrorCheck(hipMemcpy(h_debug.P_smdt, d_debug.P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  cudaErrorCheck(hipMemcpy(h_debug.dP_smdt, d_debug.dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  cudaErrorCheck(hipMemcpy(h_debug.vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
//  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void writeDebugData2File(Debug *data, std::string fileName) {
#ifdef CUDA_DEBUG
  std::ofstream fp;
  fp.open(fileName.c_str(), std::ofstream::out);
  fp.precision(16);
  //Header for file
  fp << "shell\ttheta\tvector\torder\tdegree\tg_sph_rlm\tP_smdt\tdP_smdt\tvr_rtm[0]\tvr_rtm[1]\tvr_rtm[2]\n"; 

 int idx_vr_rtm, jst, jed, m, j, l;
 int *ptr;
 for(int k=0; k<constants.nidx_rtm[0]; k++){ 
   for(int mp_rlm=constants.t_lvl+2; mp_rlm<=constants.t_lvl+3; mp_rlm++) {
     jst = data->lstack_rlm[mp_rlm-1] + 1;
     jed = data->lstack_rlm[mp_rlm];
     for(int l_rtm=0; l_rtm<constants.nidx_rtm[1]; l_rtm++) {
       for(int nd=1; nd <= constants.nvector; nd++) {
         idx_vr_rtm = (3*nd-1) + constants.ncomp*(l_rtm)*constants.istep_rtm[1] + k*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2];
         for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
           ptr = data->idx_gl_1d_rlm_j;
           m = *(ptr + (j_rlm-1 + constants.nidx_rlm[1]*2));
           l = *(ptr + (j_rlm-1 + constants.nidx_rlm[1]*1));
           j = l*(l+1) + m;
           if(m != 2) { 
             fp << k+1 << "\t" << data->g_colat_rtm[l_rtm] << "\t" << nd << "\t" << m << "\t" << l;
             fp << "\t" << data->g_sph_rlm[j_rlm-1];
             fp << "\t" << data->P_smdt[l_rtm*constants.nidx_rlm[1]+j] << "\t" << data->dP_smdt[l_rtm*constants.nidx_rlm[1]+j];
             fp << "\t" << data->vr_rtm[idx_vr_rtm-2];
             fp << "\t" << data->vr_rtm[idx_vr_rtm-1];
             fp << "\t" << data->vr_rtm[idx_vr_rtm] << "\n"; 
            }
            else {
             fp << k+1 << "\t" << data->g_colat_rtm[l_rtm] << "\t" << nd << "\t" << m << "\t" << l;
             fp << "\t" << data->g_sph_rlm[j_rlm];
             fp << "\t" << data->P_smdt[l_rtm*constants.nidx_rlm[1]+j] << "\n";
            }
         }
       }
      }
    }
  }

  fp.close();
#endif
}

void set_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
  #ifdef CUDA_TIMINGS
    cudaErrorCheck(hipDeviceSynchronize());
  #endif
}

void set_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
}

void retrieve_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
  #ifdef CUDA_TIMINGS
    cudaErrorCheck(hipDeviceSynchronize());
  #endif
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.lstack_rlm));
}

void deAllocDebugMem() {
  free(h_debug.P_smdt);
  free(h_debug.dP_smdt);
//  free(h_debug.g_sph_rlm);
  cudaErrorCheck(hipFree(d_debug.P_smdt));
  cudaErrorCheck(hipFree(d_debug.dP_smdt));
//  cudaErrorCheck(hipFree(d_debug.g_sph_rlm));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
  #ifdef CUDA_DEBUG
    hipProfilerStop();
    hipDeviceReset();
  #endif
 clockD.close();
}

void initDevConstVariables() {
  hipError_t error;
  error = hipMemcpyToSymbol(HIP_SYMBOL(devConstants), &constants, sizeof(Geometry_c), 0, hipMemcpyHostToDevice);
  cudaErrorCheck(error);
}
