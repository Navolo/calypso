#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>

Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;

double countFT=0, countBT=0;

//hipStream_t streams[32];
__constant__ Geometry_c devConstants;

//File Streams
std::ofstream *clockD;

void initgpu_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *ncomp, double *a_r_1d_rlm_r, int *lstack_rlm, double *g_colat_rtm, int *trunc_lvl, double *g_sph_rlm) {

  std::string fileName = "GPUTimings.dat";
  std::ofstream clockGPU(fileName.c_str());
  clockD = &clockGPU;  
  
  double t_0, t_1;

  constants.nidx_rtm = nidx_rtm;
  constants.nidx_rlm = nidx_rlm;
  constants.istep_rtm = istep_rtm;
  constants.istep_rlm = istep_rlm;
  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.ncomp = *ncomp;
  constants.t_lvl = *trunc_lvl; 

  #ifdef CUDA_DEBUG
    allocHostDebug(&h_debug);
    h_debug.g_colat_rtm = g_colat_rtm;
    h_debug.lstack_rlm = lstack_rlm;
    t_0 = MPI_Wtime(); 
    allocDevDebug(&d_debug);
    cudaErrorCheck(hipDeviceSynchronize());
    t_1 = MPI_Wtime();
    *clockD << "Allocation of Device Debug variables: P_smdt, dP_smdt, g_sph_rlm\t" << t_1-t_0 << std::endl;
  #endif

  t_0 = MPI_Wtime(); 
  allocMemOnGPU(lstack_rlm, a_r_1d_rlm_r, g_colat_rtm, g_sph_rlm);
  cudaErrorCheck(hipDeviceSynchronize());
  t_1 = MPI_Wtime(); 
  *clockD << "Allocation of memory for Device variables: sp_rlm, vr_rtm, g_colat_rtm, a_r_1d_rlm_r, etc\t" << t_1-t_0 << std::endl; 

  //for(unsigned int i=0; i<32; i++)       
  //  cudaErrorCheck(hipStreamCreate(&streams[i]));
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferEqual));
}

void allocMemOnGPU(int *lstack_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm, double *g_sph_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  int ncomp = constants.ncomp;
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_colat_rtm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.lstack_rlm), (constants.nidx_rtm[2]+1)*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rlm[1]*sizeof(double))); 
   
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.lstack_rlm, lstack_rlm, (constants.nidx_rtm[2]+1)*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rlm[1]*sizeof(double), hipMemcpyHostToDevice)); 
}

void allocHostDebug(Debug* h_data) {
  h_data->P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
  h_data->dP_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]);
  h_data->g_sph_rlm = (double*) malloc (sizeof(double)*constants.nidx_rlm[1]);
}
 
void allocDevDebug(Debug* d_data) {
  cudaErrorCheck(hipMalloc((void**)&(d_data->P_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(d_data->dP_smdt), sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(d_data->g_sph_rlm), sizeof(double)*constants.nidx_rlm[1]));
}

void cpyDev2Host(Debug* d_data, Debug* h_data) {
  cudaErrorCheck(hipMemcpy(d_data->P_smdt, h_data->P_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  cudaErrorCheck(hipMemcpy(d_data->dP_smdt, h_data->dP_smdt, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
  cudaErrorCheck(hipMemcpy(d_data->g_sph_rlm, h_data->g_sph_rlm, sizeof(double)*constants.nidx_rtm[1]*constants.nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void writeDebugData2File(std::ofstream *fp, Debug *data) {
#ifdef CUDA_DEBUG
  //Header for file
  *fp << "order\tdegree\tj\tshell\tidx_theta\ttheta\tg_sph_rlm\tP_smdt\tdP_smdt\tvr_rtm[0]\tvr_rtm[1]\tvr_rtm[2]\n"; 

 int idx_vr_rtm, jst, jed, m, j;
 for(int k=0; k<constants.nidx_rtm[0]; k++){ 
   for(int mp_rlm=constants.t_lvl+1; mp_rlm<=constants.t_lvl+2; mp_rlm++) {
     jst = data->lstack_rlm[mp_rlm-1] + 1;
     jed = data->lstack_rlm[mp_rlm];
     m = mp_rlm - (constants.t_lvl+1);
     for(int l=m; l<=constants.t_lvl; l++) {
        j = l*(l+1) + m; 
        for(int l_rtm=0; l<constants.nidx_rtm[1]; l++) {
          for(int nd=1; nd <= constants.nvector; nd++) {
            idx_vr_rtm = (3*nd-1) + constants.ncomp*(l_rtm)*constants.istep_rtm[1] + k*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2];
            for(int j_rlm=jst; j_rlm<=jed; j_rlm++) {
            if(m==0) 
              *fp << m << "\t" << l << "\t" << j << "\t" << k+1 << "\t" << l_rtm+1 << "\t" << data->g_colat_rtm[l_rtm] << "\t" << data->g_sph_rlm[j_rlm] << "\t" << data->P_smdt[l_rtm + constants.nidx_rtm[1]*j] << "\t" << data->dP_smdt[l_rtm + constants.nidx_rtm[1]*j] << "\t" << data->vr_rtm[idx_vr_rtm-2] << "\t" << data->vr_rtm[idx_vr_rtm-1] << "\t" << data->vr_rtm[idx_vr_rtm] << "\n"; 
            else if(m==1)
              *fp << m << "\t" << l << "\t" << j << "\t" << k+1 << "\t" << l_rtm+1 << "\t" << data->g_colat_rtm[l_rtm] << "\t" << data->g_sph_rlm[j_rlm] << "\t" << data->P_smdt[l_rtm + constants.nidx_rtm[1]*j] << "\t" << "\t" << "\t" << "\n"; 
           }
         }
       }
      }
    }
  }
#endif
}

void set_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
}

void set_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
}

void retrieve_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
    cudaErrorCheck(hipFree(deviceInput.lstack_rlm));
}

void deAllocDebugMem() {
  free(h_debug.P_smdt);
  free(h_debug.dP_smdt);
//  free(h_debug.g_sph_rlm);
  cudaErrorCheck(hipFree(d_debug.P_smdt));
  cudaErrorCheck(hipFree(d_debug.dP_smdt));
//  cudaErrorCheck(hipFree(d_debug.g_sph_rlm));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
 clockD->close();
}

void initDevConstVariables() {
  hipError_t error;
  error = hipMemcpyToSymbol(HIP_SYMBOL(devConstants), &constants, sizeof(Geometry_c), 0, hipMemcpyHostToDevice);
  cudaErrorCheck(error);
}
