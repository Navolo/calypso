#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>

Parameters_s deviceInput;
Debug h_debug, d_debug;
Geometry_c constants;

//hipStream_t streams[32];
__constant__ Geometry_c devConstants;

void initgpu_(int *nnod_rtp, int *nnod_rtm, int *nnod_rlm, int nidx_rtm[], int nidx_rlm[], int istep_rtm[], int istep_rlm[], int *ncomp, double *g_sph_rlm, double *a_r_1d_rlm_r, int *lstack_rlm, double *g_colat_rtm, int *trunc_lvl) {
  
  constants.nidx_rtm = nidx_rtm;
  constants.nidx_rlm = nidx_rlm;
  constants.istep_rtm = istep_rtm;
  constants.istep_rlm = istep_rlm;
  constants.nnod_rtp = *nnod_rtp;
  constants.nnod_rtm = *nnod_rtm;
  constants.nnod_rlm = *nnod_rlm;
  constants.ncomp = *ncomp;
  constants.t_lvl = *trunc_lvl; 

  #ifdef CUDA_DEBUG
    allocHostDebug(h_schmidt);
    h_schmidt.g_colat_rtm = g_colat_rtm;
    sllocDevDebug(d_schmidt);
  #endif

  allocMemOnGPU(g_sph_rlm, a_r_1d_rlm_r, g_colat_rtm);
  
  //for(unsigned int i=0; i<32; i++)       
  //  cudaErrorCheck(hipStreamCreate(&streams[i]));
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferEqual));
}

void allocMemOnGPU(double *g_sph_rlm, double *a_r_1d_rlm_r, double *g_colat_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  int ncomp = constants.ncomp;
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.vr_rtm), constants.nnod_rtm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.sp_rlm), constants.nnod_rlm*ncomp*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.g_sph_rlm), constants.nidx_rtm[1]*sizeof(double))); 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.a_r_1d_rlm_r), constants.nidx_rtm[0]*sizeof(double))); 
   
  cudaErrorCheck(hipMemset(deviceInput.vr_rtm, 0, constants.nnod_rtm*ncomp*sizeof(double)));
  cudaErrorCheck(hipMemset(deviceInput.sp_rlm, 0, constants.nnod_rlm*ncomp*sizeof(double)));
  cudaErrorCheck(hipMemcpy(deviceInput.g_sph_rlm, g_sph_rlm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.a_r_1d_rlm_r, a_r_1d_rlm_r , constants.nidx_rtm[0]*sizeof(double), hipMemcpyHostToDevice)); 
  cudaErrorCheck(hipMemcpy(deviceInput.g_colat_rtm, g_colat_rtm, constants.nidx_rtm[1]*sizeof(double), hipMemcpyHostToDevice)); 
}

void allocHostDebug(Debug* h_data) {
  h_data->P_smdt = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]*nidx_rlm[1]);
  h_data->g_colat_rtm = (double*) malloc (sizeof(double)*constants.nidx_rtm[1]);
}
 
void allocDevDebug(Debug* d_data) {
  cudaErrorCheck(hipMalloc((void**)&(d_data->P_smdt), sizeof(double)*constants.nidx_rtm[1]*nidx_rlm[1]));
  cudaErrorCheck(hipMalloc((void**)&(d_data->dP_smdt), sizeof(double)*constants.nidx_rtm[1]*nidx_rlm[1]));
}

void cpyDev2Host(Debug* d_data, Debug* h_data) {
  cudaErrorCheck(hipMemcpy(d_data->P_smdt, h_data->P_smdt, sizeof(double)*constants.nidx_rtm[1]*nidx_rlm[1], hipMemcpyDeviceToHost)); 
  cudaErrorCheck(hipMemcpy(d_data->dP_smdt, h_data->dP_smdt, sizeof(double)*constants.nidx_rtm[1]*nidx_rlm[1], hipMemcpyDeviceToHost)); 
}

void writeDebugData2File(std::ofstream *fp, Debug *data) {
  //Header for file
  *fp << "order\tdegree\tj\tidx_theta\ttheta\tP_smdt\tdP_smdt\n"; 

  for(int l=0; l<=constants.t_lvl; l++) {
    for(int m=0; m<=l; m++) {
      j = l*(l+1) + m; 
      for(int l_rtm=0; l<constants.nidx_rtm[1]; l++) {
        *fp << m << "\t" << l << "\t" << "\t" << j << "\t" << l_rtm << "\t" << data.g_colat_rtm[l_rtm] << "\t" << data.P_smdt[l_rtm + nidx_rtm[1]*j] << "\t" << data.dP_smdt[l_rtm + nidx_rtm[1]*j] << "\n"; 
      }
    }
  }
}

void set_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.sp_rlm, sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
}

void set_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(deviceInput.vr_rtm, vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyHostToDevice)); 
}

void retrieve_spectrum_data_(double *sp_rlm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(sp_rlm, deviceInput.sp_rlm, constants.nnod_rlm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
}

void retrieve_physical_data_(double *vr_rtm) {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMemcpy(vr_rtm, deviceInput.vr_rtm, constants.nnod_rtm*constants.ncomp*sizeof(double), hipMemcpyDeviceToHost)); 
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
    cudaErrorCheck(hipFree(deviceInput.vr_rtm));
    cudaErrorCheck(hipFree(deviceInput.sp_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_sph_rlm));
    cudaErrorCheck(hipFree(deviceInput.g_colat_rtm));
    cudaErrorCheck(hipFree(deviceInput.a_r_1d_rlm_r));
}

void deAllocDebugMem() {
  free(h_debug.P_smdt);
  free(h_debug.dP_smdt);
  cudaErrorCheck(hipFree(d_debug.P_smdt));
  cudaErrorCheck(hipFree(d_debug.dP_smdt));
}

void cleangpu_() {
  deAllocMemOnGPU();
  deAllocDebugMem();
}

void initDevConstVariables() {
  hipError_t error;
  error = hipMemcpyToSymbol(HIP_SYMBOL(devConstants), &constants, sizeof(Geometry_c), 0, hipMemcpyHostToDevice);
  cudaErrorCheck(error);
}
