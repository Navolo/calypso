#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include <math.h>

Parameters_s deviceInput;
Debug h_debug, d_debug;
//hipStream_t streams[32];

void initGpu() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  deviceInput.argc = 3; 
  deviceInput.argv = (double**) malloc (deviceInput.argc * sizeof(double*));
  // Debug: 0 = P_smdt, 1 = dp_smdt
  h_debug.argc = d_debug.argc = 2;
  h_debug.argv = (double**) malloc (h_debug.argc * sizeof(double*));
  d_debug.argv = (double**) malloc (d_debug.argc * sizeof(double*));
  allocMemOnGPU();
  //for(unsigned int i=0; i<32; i++)       
  //  cudaErrorCheck(hipStreamCreate(&streams[i]));
  cudaErrorCheck(hipDeviceSetCacheConfig(hipFuncCachePreferEqual));
}

void allocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipMalloc((void**)&(deviceInput.argv[2]), nidx_rlm[1]*sizeof(double))); 
}

void deAllocMemOnGPU() {
  // Current: 0 = vr_rtm, 1 = sp_rlm, 2 = g_sph_rlm 
  cudaErrorCheck(hipFree(deviceInput.argv[2]));
}

void finalizeGpu() {
  deAllocMemOnGPU();
}
