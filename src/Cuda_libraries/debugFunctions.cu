#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "legendre_poly.h"
#include <math.h>
#include <iostream>

void check_bwd_trans_cuda_(int *my_rank, double *vr_rtm, double *P_jl, double *dP_jl) {
  #if defined(CUDA_DEBUG) || defined(CHECK_SCHMIDT_OTF)
    std::ofstream schmidt, field_vec, field_slr;
    schmidt.open("cuda_schimdt.log");
    field_vec.open("cuda_field_vectors.log");
    field_slr.open("cuda_field_scalars.log");
      
    schmidt << "order\t degree\t P_smdt \t P_smdt_cuda\t dP_smdt \t dP_smdt_cuda\n";

    double error=0, eps = 1E-7;
    int jst, jed, ip_rtm, in_rtm, pos, m, l, mn_rlm;

    for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++){
      jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
      jed = h_debug.lstack_rlm[mp_rlm];
      for(int l_rtm=1; l_rtm<=constants.nidx_rtm[1]; l_rtm++) {
        for(int j_rlm=jst; j_rlm <=jed; j_rlm++) {
          m = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1]*2 + j_rlm-1];
          l = h_debug.idx_gl_1d_rlm_j[constants.nidx_rlm[1] + j_rlm-1];
          pos = (l_rtm-1)*constants.nidx_rlm[1] + l*(l+1) + m;
          schmidt << m << "\t" <<  l << "\t" << P_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1] << "\t" << h_debug.P_smdt[pos]<< "\t" << dP_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1] << "\t" << h_debug.dP_smdt[pos] << "\n"; 
        }
      }
    }
   schmidt.close();

   field_vec << "shell\tmeridian\tmp_rlm\tvector_index\t vr_rtm_cu[0]\t vr_rtm[0] \t vr_rtm_cu[1] \t vr_rtm[1] \t vr_rtm_cu[2] \t vr_rtm[2] \t vr_rtm_n_cu[0] \t vr_rtm_n[0] \t vr_Rtm_n_cu[1] \t vr_rtm_n[1]\n";
   field_slr << "shell\tmeridian\tmp_rlm\tscalar\t vr_rtm_cu[0]\t vr_rtm[0] \n";

    for(int k=1; k<=constants.nidx_rtm[0]; k++) {
      for(int mp_rlm=1; mp_rlm<=constants.nidx_rtm[2]; mp_rlm++) {
        jst = h_debug.lstack_rlm[mp_rlm-1] + 1; 
        jed = h_debug.lstack_rlm[mp_rlm];
        mn_rlm = constants.nidx_rtm[2] - mp_rlm + 1;
        for(int l_rtm=1; l_rtm <=constants.nidx_rtm[1]; l_rtm++) {
          for(int nd=1; nd<=constants.nvector; nd++) {
            ip_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            in_rtm = 3*nd + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mn_rlm-1)*constants.istep_rtm[2]) - 1;
            field_vec << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t"<< h_debug.vr_rtm[ip_rtm-2] << "\t" << vr_rtm[ip_rtm-2] << "\t" << h_debug.vr_rtm[ip_rtm-1] << "\t" << vr_rtm[ip_rtm-1] << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\t" << h_debug.vr_rtm[in_rtm-1] << "\t" << vr_rtm[in_rtm-1] << h_debug.vr_rtm[in_rtm] << "\t" << vr_rtm[in_rtm] << "\n";
          }
          for(int nd=1; nd<=constants.nscalar; nd++) {
            ip_rtm = nd + 3*constants.nvector + constants.ncomp*((l_rtm-1)*constants.istep_rtm[1] + (k-1)*constants.istep_rtm[0] + (mp_rlm-1)*constants.istep_rtm[2]) - 1;
            field_slr << k << "\t" << l_rtm << "\t" << mp_rlm << "\t" << nd << "\t" << h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\n";
          }
/*              error = abs(h_debug.P_smdt[pos] - P_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1]); 
              if(error > eps) { 
                data_log << "Error of " << error << "at P_smdt for l=" << l<< " and m=" << m <<std::endl; 
                //exit(-1);
              }
              error = abs(h_debug.dP_smdt[pos] - dP_jl[constants.nidx_rlm[1]*(l_rtm-1) + j_rlm-1]); 
              if(error > eps) { 
                data_log << "Error of " << error << "at dP_smdt for l=" << l<< " and m=" << m <<std::endl; 
                //exit(-1);
              }
              error = abs(h_debug.vr_rtm[ip_rtm-2] - vr_rtm[ip_rtm-2]); 
              if(error > eps) { 
                data_log << "Error of " << error << "at vr_rtm first comp for l=" << l<< " and m=" << m <<std::endl; 
                //exit(-1);
              }
              error = abs(h_debug.vr_rtm[ip_rtm-1] - vr_rtm[ip_rtm-1]); 
              if(error > eps) { 
                data_log << "Error of " << error << "at vr_rtm mid comp for l=" << l<< " and m=" << m <<std::endl; 
                //exit(-1);
              }
              error = abs(h_debug.vr_rtm[ip_rtm] - vr_rtm[ip_rtm]); 
              if(error > eps) { 
                data_log << "Error of " << error << "at vr_rtm last comp for l=" << l<< " and m=" << m <<std::endl; 
                //exit(-1);
              }
              vr_rtm_log<<h_debug.vr_rtm[in_rtm-1] << "\t" << vr_rtm[in_rtm-1] << "\n";
              vr_rtm_log<<h_debug.vr_rtm[in_rtm] << "\t" << vr_rtm[in_rtm] << "\n";
              vr_rtm_log<<h_debug.vr_rtm[ip_rtm-2] << "\t" << vr_rtm[ip_rtm-2] << "\n";
              vr_rtm_log<<h_debug.vr_rtm[ip_rtm-1] << "\t" << vr_rtm[ip_rtm-1] << "\n";
              vr_rtm_log<<h_debug.vr_rtm[ip_rtm] << "\t" << vr_rtm[ip_rtm] << "\n";
            */
        }
      }
    }
   
   field_vec.close();
   field_slr.close();
   exit(-1);
  #endif
}
